#include "hip/hip_runtime.h"
#include "gpu_shared.h"

__global__ void normalizationGPU(long* vec_time, double* vec_normal_time, long max_value, long size) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vec_normal_time[idx] = (double)vec_time[idx] / (double)max_value; // Нормализация
    }
}

extern "C" double* normalization( long* vec_time,long vec_size, long max_value, int amount_blocks, int amount_threads) {
    long size = vec_size;
    double *vec_normal_time = (double*)malloc(size * sizeof(double));
    long *d_vec_time;
    double *d_vec_normal_time;
    hipMalloc(&d_vec_time, size * sizeof(long));
    hipMalloc(&d_vec_normal_time, size * sizeof(double));
    hipMemcpy(d_vec_time, vec_time, size * sizeof(long),hipMemcpyHostToDevice);
    
    normalizationGPU<<<amount_threads, amount_blocks>>>(d_vec_time,d_vec_normal_time,max_value, size);

    hipMemcpy(vec_normal_time, d_vec_normal_time, size * sizeof(double),hipMemcpyDeviceToHost);
    hipFree(d_vec_time);
    hipFree(d_vec_normal_time);
    return vec_normal_time;
}


